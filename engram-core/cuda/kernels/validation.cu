#include "hip/hip_runtime.h"
#include "../common.cuh"
#include <stdio.h>

// Minimal validation kernel to verify CUDA toolchain integration
// This kernel performs a trivial computation to prove that:
// 1. CUDA code compiles correctly
// 2. Kernels can be launched from Rust
// 3. Data can be transferred between host and device
// 4. Results are computed correctly on GPU

__global__ void hello_kernel(int* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        // Simple computation: output[i] = i * 2
        output[idx] = idx * 2;
    }
}

// Test kernel with 768-dimensional vector operations
__global__ void vector_test_kernel(const float* input, float* output, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < count) {
        // Simple vector operation: accumulate first 8 elements
        float sum = 0.0f;
        const float* vec = input + idx * EMBEDDING_DIM;

        #pragma unroll
        for (int i = 0; i < 8; i++) {
            sum += vec[i];
        }

        output[idx] = sum;
    }
}

// External C interface for Rust FFI
extern "C" {

// Validate that CUDA environment is functional
// Returns:
//   0: Success - GPU is available and functional
//  -1: hipGetDeviceCount failed
//  -2: No CUDA devices found
//  -3: hipMalloc failed (GPU memory allocation error)
int cuda_validate_environment() {
    int device_count;
    hipError_t err = hipGetDeviceCount(&device_count);

    if (err != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n",
                hipGetErrorString(err));
        return -1;
    }

    if (device_count == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        return -2;
    }

    // Try allocating and freeing a small amount of memory
    void* test_ptr;
    err = hipMalloc(&test_ptr, 1024);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc test failed: %s\n",
                hipGetErrorString(err));
        return -3;
    }

    hipFree(test_ptr);
    return 0;
}

// Test kernel launch with trivial computation
// Arguments:
//   h_output: Host array to receive results (must be allocated by caller)
//   size: Number of elements in output array
// Returns:
//   0: Success - kernel launched and completed correctly
//  -1: hipMalloc failed
//  -2: hipMemcpy failed
int cuda_test_kernel_launch(int* h_output, int size) {
    int* d_output;

    // Allocate device memory
    hipError_t err = hipMalloc((void**)&d_output, size * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Launch kernel with appropriate grid dimensions
    int block_size = DEFAULT_BLOCK_SIZE;
    int grid_size;
    compute_grid_dimensions(size, block_size, grid_size);

    hello_kernel<<<grid_size, block_size>>>(d_output, size);

    // Wait for kernel to complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n",
                hipGetErrorString(err));
        hipFree(d_output);
        return -2;
    }

    // Copy results back to host
    err = hipMemcpy(h_output, d_output, size * sizeof(int),
                     hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
        hipFree(d_output);
        return -2;
    }

    hipFree(d_output);
    return 0;
}

// Test vector operations kernel
// Arguments:
//   h_input: Host array of 768-dimensional vectors (count * 768 floats)
//   h_output: Host array to receive results (count floats)
//   count: Number of vectors
// Returns:
//   0: Success
//  -1: hipMalloc failed
//  -2: hipMemcpy failed
int cuda_test_vector_kernel(const float* h_input, float* h_output, int count) {
    float *d_input, *d_output;
    size_t input_size = count * EMBEDDING_DIM * sizeof(float);
    size_t output_size = count * sizeof(float);

    // Allocate device memory
    hipError_t err = hipMalloc((void**)&d_input, input_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc input failed: %s\n",
                hipGetErrorString(err));
        return -1;
    }

    err = hipMalloc((void**)&d_output, output_size);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc output failed: %s\n",
                hipGetErrorString(err));
        hipFree(d_input);
        return -1;
    }

    // Copy input to device
    err = hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy input failed: %s\n",
                hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return -2;
    }

    // Launch kernel
    int block_size = DEFAULT_BLOCK_SIZE;
    int grid_size;
    compute_grid_dimensions(count, block_size, grid_size);

    vector_test_kernel<<<grid_size, block_size>>>(d_input, d_output, count);

    // Wait for kernel to complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n",
                hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return -2;
    }

    // Copy results back
    err = hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy output failed: %s\n",
                hipGetErrorString(err));
        hipFree(d_input);
        hipFree(d_output);
        return -2;
    }

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}

} // extern "C"
